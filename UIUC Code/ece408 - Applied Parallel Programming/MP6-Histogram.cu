#include "hip/hip_runtime.h"
// Histogram Equalization

#include <wb.h>
#define HISTOGRAM_LENGTH 256

__global__ void floatToChar(float* floot, unsigned char* chor, int width, int height, int imageChannels){
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  if(i < width*height*imageChannels){
    chor[i] = (unsigned char)(255*floot[i]);
  }
}
//change for loops to if statements

__global__ void RGBtoGray(unsigned char* ucharImage, unsigned char* grayImage, int width,int height, int imageChannels){
  int x = threadIdx.x + blockIdx.x*blockDim.x;
  int y = threadIdx.y + blockIdx.y*blockDim.y;
  if(threadIdx.x + blockIdx.x*blockDim.x < width*height*imageChannels){
    int idx = y*width+x;
    float r = ucharImage[3*idx];
    float g = ucharImage[3*idx+1];
    float b = ucharImage[3*idx+2];
    grayImage[idx] = (unsigned char)(0.21*r + 0.71*g + 0.07*b);

  }
}


//@@ insert code here
__global__ void histo_kernel(unsigned char *buffer, int size, unsigned int *histo){
  __shared__ unsigned int private_histo[256];
  if(threadIdx.x < 256) private_histo[threadIdx.x] = 0;
  __syncthreads();
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int stride = blockDim.x * gridDim.x;
  while(i<size){
    atomicAdd(&(private_histo[buffer[i]]),1);
    i += stride;
  }
  __syncthreads();
  if(threadIdx.x < 256){
    atomicAdd(&(histo[threadIdx.x]), private_histo[threadIdx.x]);
  }
}

__global__ void scan(unsigned int *input, float *output, int len, int width, int height) {
  __shared__ float T[HISTOGRAM_LENGTH];
  int i = 2*blockIdx.x*blockDim.x + threadIdx.x;
  if(i<len)
    T[threadIdx.x] = input[i];
  if (i+blockDim.x < len)
    T[threadIdx.x+blockDim.x]=input[i+blockDim.x];

  for (unsigned int stride = 1; stride <= blockDim.x;stride *= 2){
    __syncthreads();
    int index = (threadIdx.x+1)* 2* stride -1;
    if(index<HISTOGRAM_LENGTH)
      T[index] += T[index-stride];
  }
  for (int stride = 64;stride>0; stride /= 2){
    __syncthreads();
    int index = (threadIdx.x+1)*stride*2 - 1;
    if(index + stride < HISTOGRAM_LENGTH)
      T[index + stride] += T[index];
  }

  __syncthreads();
  if(i<len)
    output[i] = T[threadIdx.x]/(1.0*width*height);
  if(i+blockDim.x<len)
    output[i+blockDim.x] = T[threadIdx.x+blockDim.x]/(1.0*width*height);

}



__global__ void histEqual(float* cdf, unsigned char* out, int width, int height, int imageChannels){
  //cdf = in
  int idx = threadIdx.x + blockIdx.x*blockDim.x;
  //float val = out[idx];
  //float cdfmin = cdf[0];
  if(idx < width*height*imageChannels){
    out[idx] = (unsigned char) (min(max(255.0*(cdf[out[idx]]-cdf[0])/(1.0-cdf[0]),0.0),255.0));
  }
}


__global__ void charToFloat(unsigned char* in, float* out, int width, int height, int imageChannels){
  int i = threadIdx.x + blockIdx.x*blockDim.x;

  if(i<width*height*imageChannels){
    out[i] = (float) (in[i]/255.0);
  }
}




int main(int argc, char **argv) {
  wbArg_t args;
  int imageWidth;
  int imageHeight;
  int imageChannels;
  int imageWH;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  const char *inputImageFile;

  //@@ Insert more code here
  unsigned char* uchar;
  float* floot;
  unsigned int* kurnel;
  unsigned char* gray;
  float* scan1;

  args = wbArg_read(argc, argv); /* parse the input arguments */

  inputImageFile = wbArg_getInputFile(args, 0);

  wbTime_start(Generic, "Importing data and creating memory on host");
  inputImage = wbImport(inputImageFile);
  imageWidth = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  imageChannels = wbImage_getChannels(inputImage);
  outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);
  hostInputImageData = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);
  wbTime_stop(Generic, "Importing data and creating memory on host");
  imageWH = imageWidth*imageHeight;
  //@@ insert code here
  hipMalloc((void**)&uchar, imageWH*imageChannels*sizeof(unsigned char));
  hipMalloc((void**)&floot, imageWH*imageChannels*sizeof(float));
  hipMalloc((void**)&kurnel, HISTOGRAM_LENGTH*sizeof(unsigned int));
  hipMalloc((void**)&gray, imageWH*sizeof(unsigned char));
  hipMalloc((void**)&scan1, HISTOGRAM_LENGTH*sizeof(float));

  hipMemcpy(floot, hostInputImageData, imageWH*imageChannels*sizeof(float),hipMemcpyHostToDevice);
  int twoLen = HISTOGRAM_LENGTH*2;
  //1
  dim3 dimGridChar(ceil(imageWH*imageChannels*1.0/twoLen),1,1);
  dim3 dimBlockChar(twoLen,1,1);
  floatToChar<<<dimGridChar,dimBlockChar>>>(floot, uchar, imageWidth, imageHeight, imageChannels);
  hipDeviceSynchronize();
  //2
  dim3 dimGridGray(ceil(imageWH*1.0/twoLen),1,1);
  dim3 dimBlockGray(twoLen,1,1);
  RGBtoGray<<<dimGridGray,dimBlockGray>>>(uchar, gray, imageWidth, imageHeight, imageChannels);
  hipDeviceSynchronize();
  //3
  dim3 dimGridKurnel(ceil(imageWH*1.0/HISTOGRAM_LENGTH),1,1);
  dim3 dimBlockKurnel(HISTOGRAM_LENGTH,1,1);
  histo_kernel<<<dimGridKurnel,dimBlockKurnel>>>(gray,imageWH,kurnel);
  hipDeviceSynchronize();
  //4
  dim3 dimGridScan(1,1,1);
  dim3 dimBlockScan(128,1,1);
  scan<<<dimGridScan,dimBlockScan>>>(kurnel,scan1,HISTOGRAM_LENGTH,imageWidth,imageHeight);
  hipDeviceSynchronize();
  //5
  dim3 dimGridEqual(ceil(imageWH*imageChannels*1.0/twoLen),1,1);
  dim3 dimBlockEqual(twoLen,1,1);
  histEqual<<<dimGridEqual, dimBlockEqual>>>(scan1,uchar,imageWidth,imageHeight,imageChannels);
  hipDeviceSynchronize();
  //6
  dim3 dimGrid(ceil(imageWH*imageChannels*1.0/twoLen),1,1);
  dim3 dimBlock(twoLen,1,1);
  charToFloat<<<dimGrid,dimBlock>>>(uchar, floot, imageWidth, imageHeight, imageChannels);
  hipDeviceSynchronize();

  hipMemcpy(hostOutputImageData, floot,imageWH*imageChannels*sizeof(float),hipMemcpyDeviceToHost);
  wbImage_setData(outputImage,hostOutputImageData);
  wbSolution(args, outputImage);
  hipFree(hostInputImageData);
  hipFree(hostOutputImageData);
  hipFree(uchar);
  hipFree(floot);
  hipFree(kurnel);
  hipFree(gray);
  hipFree(scan1);

  //@@ insert code here


  return 0;
}
