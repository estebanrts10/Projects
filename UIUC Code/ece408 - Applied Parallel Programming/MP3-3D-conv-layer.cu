#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define TILE_WIDTH 4
#define MASK_WIDTH 3
//@@ Define constant memory for device kernel here
__constant__ float kernelCpy[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];

__global__ void conv3d(float *input, float *output, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float tile[TILE_WIDTH+MASK_WIDTH-1][TILE_WIDTH+MASK_WIDTH-1][TILE_WIDTH+MASK_WIDTH-1];
  int radius = MASK_WIDTH / 2;
  //int start = i-radius;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int Width = x_size;
  int Height = y_size;
  int Depth = z_size;
  int dep_o = blockIdx.z * TILE_WIDTH + tz;
  int row_o = blockIdx.y * TILE_WIDTH + ty;
  int col_o = blockIdx.x * TILE_WIDTH + tx;

  int dep_i = dep_o - radius;
  int row_i = row_o - radius; //mask_width / 2
  int col_i = col_o - radius; //radius

  //create width variable

  if((row_i >= 0) && (row_i < Height) && (col_i >= 0) &&
     (col_i < Width) && (dep_i >=0) && (dep_i < Depth)){
    tile[tz][ty][tx] = input[col_i + row_i*Width + dep_i*Width*Height];
    //change this input index to account for 3d

  }
  else{
    tile[tz][ty][tx] = 0.0f;
  }

  __syncthreads();
  float Pvalue = 0.0f;
  if(ty < TILE_WIDTH && tx < TILE_WIDTH && tz < TILE_WIDTH){
    for(int z = 0; z < MASK_WIDTH; z++){
      for(int y = 0; y < MASK_WIDTH; y++){
        for(int x = 0; x < MASK_WIDTH; x++){
          Pvalue += kernelCpy[z][y][x] * tile[z+tz][y+ty][x+tx];
        }
      }
    }
    if(row_o < Height && col_o < Width && dep_o < Depth){
      output[dep_o*Width*Height + row_o*Width + col_o] = Pvalue;
    }
  }

}

int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  float *deviceInput;
  float *deviceOutput;

  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ Allocate GPU memory here
  hipMalloc((void**) &deviceInput, (inputLength-3)*sizeof(float));
  hipMalloc((void**) &deviceOutput, (inputLength-3)*sizeof(float));
  // Recall that inputLength is 3 elements longer than the input data
  // because the first  three elements were the dimensions
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  //@@ Copy input and kernel to GPU here

  hipMemcpy(deviceInput, hostInput+3, (inputLength-3)*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(kernelCpy), hostKernel, kernelLength*sizeof(float));
  //have to copy kernel to gpu still


  // Recall that the first three elements of hostInput are dimensions and
  // do
  // not need to be copied to the gpu
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ Initialize grid and block dimensions here
  dim3 dimGrid(ceil(x_size/(TILE_WIDTH*1.0)),ceil(y_size/(TILE_WIDTH*1.0)),ceil(z_size/(TILE_WIDTH*1.0)));
  dim3 dimBlock(TILE_WIDTH+MASK_WIDTH-1,TILE_WIDTH+MASK_WIDTH-1,TILE_WIDTH+MASK_WIDTH-1);

  //@@ Launch the GPU kernel here
  conv3d<<<dimGrid,dimBlock>>>(deviceInput,deviceOutput,z_size,y_size,x_size);

  hipDeviceSynchronize();
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ Copy the device memory back to the host here
  hipMemcpy(hostOutput+3, deviceOutput, (inputLength-3)*sizeof(float), hipMemcpyDeviceToHost);
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  // Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}
